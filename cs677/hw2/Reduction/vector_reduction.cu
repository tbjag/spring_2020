#include "hip/hip_runtime.h"
/*
* Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
*
* NOTICE TO USER:   
*
* This source code is subject to NVIDIA ownership rights under U.S. and 
* international Copyright laws.  
*
* This software and the information contained herein is PROPRIETARY and 
* CONFIDENTIAL to NVIDIA and is being provided under the terms and 
* conditions of a Non-Disclosure Agreement.  Any reproduction or 
* disclosure to any third party without the express written consent of 
* NVIDIA is prohibited.     
*
* NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
* CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
* IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
* REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
* MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
* IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
* OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
* OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
* OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
* OR PERFORMANCE OF THIS SOURCE CODE.  
*
* U.S. Government End Users.  This source code is a "commercial item" as 
* that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
* "commercial computer software" and "commercial computer software 
* documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
* and is provided to the U.S. Government only as a commercial end item.  
* Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
* 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
* source code with only those rights set forth herein.
*/

#ifdef _WIN32
#define NOMINMAX 
#endif

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <float.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

// int readFile(filename:str, data: array);
#include "file_io.h"

// includes, kernels
#include <vector_reduction_kernel.cu>
//#include <vector_reduction_kernel_adv.cu>

// For simplicity, just to get the idea in this MP, we're fixing the problem size to 512 elements.
#define NUM_ELEMENTS 512
#define BLOCK_SIZE 32

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);

float computeOnDevice(float* h_data, int array_mem_size);

extern "C" 
void computeGold( float* reference, float* idata, const unsigned int len);

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int 
main( int argc, char** argv) 
{
    runTest( argc, argv);
    return EXIT_SUCCESS;
}

////////////////////////////////////////////////////////////////////////////////
//! Run naive scan test
////////////////////////////////////////////////////////////////////////////////
void
runTest( int argc, char** argv) 
{
    int num_elements = NUM_ELEMENTS;
    int errorM = 0;

    const unsigned int array_mem_size = sizeof( float) * num_elements;

    // allocate host memory to store the input data
    float* h_data = (float*) malloc( array_mem_size);

    // * No arguments: Randomly generate input data and compare against the 
    //   host's result.
    // * One argument: Read the input data array from the given file.
    switch(argc-1)
    {      
        
        case 1:  // One Argument
            errorM = readFile(argv[1], h_data);
            if(errorM != 1)
            {
                printf("Error reading input file!\n");
                exit(1);
            }
        break;
        
        
        default:  // No Arguments or one argument
            // initialize the input data on the host to be integer values
            // between 0 and 1000
            for( unsigned int i = 0; i < num_elements; ++i) 
            {
                h_data[i] = floorf(1000*(rand()/(float)RAND_MAX));
            }
        break;  
    }
    // compute reference solution
    float reference = 0.0f;  
    computeGold(&reference , h_data, num_elements);
    
    // **===-------- Modify the body of this function -----------===**
    float result = computeOnDevice(h_data, num_elements);
    // **===-----------------------------------------------------------===**


    // We can use an epsilon of 0 since values are integral and in a range 
    // that can be exactly represented
    float epsilon = 0.0f;
    unsigned int result_regtest = (abs(result - reference) <= epsilon);
    printf( "Test %s\n", (1 == result_regtest) ? "PASSED" : "FAILED");
    printf( "device: %f  host: %f\n", result, reference);
    // cleanup memory
    free( h_data);
}

// **===----------------- Modify this function ---------------------===**
// Take h_data from host, copies it to device, setup grid and thread 
// dimentions, executes kernel function, and copy result of scan back
// to h_data.
// Note: float* h_data is both the input and the output of this function.
float computeOnDevice(float* h_data, int num_elements)
{
	// num is zero or 1 just return solution
	if(num_elements == 0)
		return 0.0f;
	else if (num_elements == 1)
		return h_data[0];
	else if (num_elements % 2 != 0)
		num_elements += 1; //will just have a 0 at the end
	
	// declare device vector 
	float *d_data;
	
	// calc number of bytes
	size_t bytes = num_elements * sizeof(float);
	
	// malloc on device
	hipMalloc(&d_data, bytes);
	
	// copy data to device
	hipMemcpy( d_data, h_data, bytes, hipMemcpyHostToDevice);
	
	int block_size, grid_size, half_elements;
	
	//calc block size and grid size 
	//block_size = 256;
	//grid_size = (int)ceil((num_elements/2)/(float)block_size);
	//block_size = num_elements/2;
	//grid_size = (int)ceil();
	half_elements = num_elements/2;
	
	printf("%d\n", block_size);
	//send to appropriate function 
	if(num_elements <= 512){
		//appropriate block size 
		block_size = (half_elements) % 32 == 0 ? half_elements : half_elements + (BLOCK_SIZE - half_elements%BLOCK_SIZE); 
		reduction<<<1, block_size >>>(d_data, num_elements);
	} else{
		//work on this
		block_size = 256;
		//grid_size = (int)ceil((num_elements/2)/(float)block_size);
		//reduction_adv<<<grid_size, block_size >>>(d_data, num_elements);
	}

	// Copy result back to host
	hipMemcpy( h_data, d_data, bytes, hipMemcpyDeviceToHost );
	
	// print out result
	for(int i = 0; i < NUM_ELEMENTS; i++){
		printf("%lf ", h_data[i]);
	}
	printf("\n");
	
	// release memory
	hipFree(d_data);
	
	// return single point
	return h_data[0];

}
     
