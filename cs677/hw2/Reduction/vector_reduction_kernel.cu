/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

#ifndef _SCAN_NAIVE_KERNEL_H_
#define _SCAN_NAIVE_KERNEL_H_

#define NUM_ELEMENTS 512


// **===----------------- MP3 - Modify this function ---------------------===**
//! @param g_idata  input data in global memory
//                  result is expected in index 0 of g_idata
//! @param n        input number of elements to scan from input data
// **===------------------------------------------------------------------===**
__global__ void reduction(float *g_data, int n)
{
	__shared__ float partial_sum[NUM_ELEMENTS/2];
	
	//find id
	unsigned int t = threadIdx.x;
	
	//load from global into shared mem, do the first computation
	partial_sum[t] = g_data[t] + g_data[t + n/2];
	
	for(unsigned int stride = blockDim.x/2; stride >= 1; stride >>= 1){
		__syncthreads();
		if(t < stride)
			partial_sum[t] += partial_sum[t+stride];
	}

	//put result into global
	if(t < 1)
		g_data[t] = partial_sum[t];
}

__global__ void reduction_adv(float *g_data, int n)
{
	__shared__ float partial_sum[NUM_ELEMENTS/2];
	
	//find id
	unsigned int t = threadIdx.x;
	
	//load from global into shared mem, do the first computation
	partial_sum[t] = g_data[t] + g_data[t + n/2];
	
	for(unsigned int stride = blockDim.x/2; stride >= 1; stride >>= 1){
		__syncthreads();
		if(t < stride)
			partial_sum[t] += partial_sum[t+stride];
	}

	//put result into global
	if(t < 1)
		g_data[t] = partial_sum[t];
}

#endif // #ifndef _SCAN_NAIVE_KERNEL_H_
