#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2006 NVIDIA Corporation.  All rights reserved.
 *
 * NOTICE TO USER:   
 *
 * This source code is subject to NVIDIA ownership rights under U.S. and 
 * international Copyright laws.  
 *
 * This software and the information contained herein is PROPRIETARY and 
 * CONFIDENTIAL to NVIDIA and is being provided under the terms and 
 * conditions of a Non-Disclosure Agreement.  Any reproduction or 
 * disclosure to any third party without the express written consent of 
 * NVIDIA is prohibited.     
 *
 * NVIDIA MAKES NO REPRESENTATION ABOUT THE SUITABILITY OF THIS SOURCE 
 * CODE FOR ANY PURPOSE.  IT IS PROVIDED "AS IS" WITHOUT EXPRESS OR 
 * IMPLIED WARRANTY OF ANY KIND.  NVIDIA DISCLAIMS ALL WARRANTIES WITH 
 * REGARD TO THIS SOURCE CODE, INCLUDING ALL IMPLIED WARRANTIES OF 
 * MERCHANTABILITY, NONINFRINGEMENT, AND FITNESS FOR A PARTICULAR PURPOSE.   
 * IN NO EVENT SHALL NVIDIA BE LIABLE FOR ANY SPECIAL, INDIRECT, INCIDENTAL, 
 * OR CONSEQUENTIAL DAMAGES, OR ANY DAMAGES WHATSOEVER RESULTING FROM LOSS 
 * OF USE, DATA OR PROFITS, WHETHER IN AN ACTION OF CONTRACT, NEGLIGENCE 
 * OR OTHER TORTIOUS ACTION, ARISING OUT OF OR IN CONNECTION WITH THE USE 
 * OR PERFORMANCE OF THIS SOURCE CODE.  
 *
 * U.S. Government End Users.  This source code is a "commercial item" as 
 * that term is defined at 48 C.F.R. 2.101 (OCT 1995), consisting  of 
 * "commercial computer software" and "commercial computer software 
 * documentation" as such terms are used in 48 C.F.R. 12.212 (SEPT 1995) 
 * and is provided to the U.S. Government only as a commercial end item.  
 * Consistent with 48 C.F.R.12.212 and 48 C.F.R. 227.7202-1 through 
 * 227.7202-4 (JUNE 1995), all U.S. Government End Users acquire the 
 * source code with only those rights set forth herein.
 */

/* Matrix multiplication: C = A * B.
 * Device code.
 */

#ifndef _MATRIXMUL_KERNEL_H_
#define _MATRIXMUL_KERNEL_H_

#include <stdio.h>
#include "matrixmul.h"

//change tile width
#define TILE_WIDTH 256

////////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_idata  input data in global memory
//! @param g_odata  output data in global memory
////////////////////////////////////////////////////////////////////////////////
// Matrix multiplication kernel thread specification
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	__shared__ float m_local_shared[TILE_WIDTH][TILE_WIDTH];
	__shared__ float n_local_shared[TILE_WIDTH][TILE_WIDTH];
	
	int bx = blockIdx.x; int by = blockIdx.y; 
	int tx = threadIdx x; int ty = threadIdx.y;
	
	//identifies which tile in section you are working in
	int row = by*TILE_WIDTH + ty;
	int col = bx*TILE_WIDTH + tx;
	
	int intermediate_val = 0;
	int num_of_tiles = (int)ceil((float)n.height/TILE_WIDTH);//go above limit? m.width = n.height 
	
	for(int count = 0; count < num_of_tiles; count++){
		m_local_shared[ty][tx] = M[row*M.width + (count*TILE_WIDTH + tx)]; //find the right section for m!
		n_local_shared[ty][tx] = N[(count*TILE_WIDTH + ty)*N.width + col]; //find the right section for n!
		
		__syncthreads();
		
		for(int k =0; k < TILE_WIDTH; k++)
			intermediate_val += m_local_shared[ty][k] * n_local_shared[k][tx];
		
		__syncthreads();
	}
	
	P[row*M.width + col] = intermediate_val;
}

#endif // #ifndef _MATRIXMUL_KERNEL_H_
