//optimization homework #4 cs 677 Theodore Jagodits


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "string.h"
#include <iostream>

#define DEFAULT_SIZE 128
#define DEFAULT_WIDTH 128
#define DEFAULT_HEIGHT 128
#define TILE_SIZE 16

__global__ void unknown_algo(float *inp1, float *inp2, float *result, int width, int height){
	// make shared
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float temp, k_loop_temp, inpt1_s;
	temp = 0.0f;
	for(int j = 0; j < height; j++){
		temp += inp2[id * width + j];
		result[id * width + j] = temp;
		k_loop_temp = 0.0f;
		inpt1_s = inp1[j];
		for(int k = 0; k < height; k++){
			k_loop_temp += inpt1_s * inp1[k];
		}
		//speed up here
		result[id * width + j] += k_loop_temp;
	}
}

int main( int argc, char **argv ){
	int size = DEFAULT_SIZE;
	int width = DEFAULT_WIDTH;
	int height = DEFAULT_HEIGHT;
	if(argc == 3){
		//size = atoi(argv[1]);
		width = atoi(argv[1]);
		height = atoi(argv[2]);
	}
	
	//create vars
	int input1_bytes = height * sizeof(float);
	int num_bytes = width * height * sizeof(float);
	
	//event timers
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//malloc device
	float *d_input1 = (float *) malloc(input1_bytes);
	float *d_input2 = (float *) malloc(num_bytes);
	float *d_result = (float *) malloc(num_bytes);
	
	//malloc host
	float *h_input1 = (float *) malloc(input1_bytes);
	float *h_input2 = (float *) malloc(num_bytes);
	float *h_result = (float *) malloc(num_bytes);
	
	//cuda malloc
	hipMalloc(&d_input1, input1_bytes);
	hipMalloc(&d_input2, num_bytes);
	hipMalloc(&d_result, num_bytes);
	
	//put in data
	for(int o = 0; o < width; o++){
        h_input1[o] = 1;
        for(int p = 0; p < height; p++){
            h_input2[width * o + p] = 1;
        }
    }

	//copy over memory
	hipMemcpy(d_input1, h_input1, input1_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_input2, h_input2, num_bytes, hipMemcpyHostToDevice);
	
	//declare block and grid size for kernel
	int block_size = 128;
	int grid_size = (int)ceil((float)width/block_size);
	
	//start timer 
	hipEventRecord(start);
	
	//run kernel
	unknown_algo<<< grid_size, block_size >>> (d_input1, d_input2, d_result, width, height);
	
	//end timer
	hipEventRecord(stop);
	
	// Copy result back to host
	hipMemcpy(h_result, d_result, num_bytes, hipMemcpyDeviceToHost);
	
	//synchronize https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	//print output
	for(int o = 0; o < width; o++){
        for(int p = 0; p < height; p++){
            printf("%d ", (int)h_result[o*width + p]);
        }
		printf("\n");
    }
	printf("time for execution: %lf ms\n", milliseconds);
	
	//free all vars
	free(h_input1);
	free(h_input2);
	free(h_result);
	hipFree(d_input1);
	hipFree(d_input2);
	hipFree(d_result);
	
	return 0;
}