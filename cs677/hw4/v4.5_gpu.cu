#include "hip/hip_runtime.h"
//optimization homework #4 cs 677 Theodore Jagodits
#include <time.h>
#include <stdio.h>
#include <stdlib.h>
#include "string.h"
#include <iostream>

#define DEFAULT_WIDTH 128
#define DEFAULT_HEIGHT 128
#define TILE_SIZE 32

__global__ void unknown_algo(float *inp1, float *inp2, float *result, int width, int height){
	//get row col idx
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	//check for bounds here
	if(row < width && col < height){
		//make vars for temp, k_temp, input1[ty]
		float temp = 0.0f, k_loop_t = 0.0f, input1_store = inp1[col];
		//find the first part of the result
		for(int count = 0; count <= col; count++){
			temp += inp2[row*width + count];
		}
		//store result
		result[row*width + col] = 5;
		//find inpt1 of result
			k_loop_t += input1_store * inp1[k];
		for(int k = 0; k < height; k++){
		}
		//add to answer
		result[row*width + col] += k_loop_t;	
	}
}

void cpu_v(float *inp1, float *inp2, float *result, int width, int height, float *temp){
	for(int i = 0 ; i < width; i++){
		temp[i] = 0.0f;
		for(int j = 0; j < height; j++){
			temp[i] += inp2[i*width + j];
			result[i*width + j] = temp[i];
			for(int k = 0; k < height; k++){
				result[i*width + j] += inp1[j] * inp1[k];
			}
		}
	}
}

int compare_res(float *d_res, float *h_res, int width, int height){
	int check = 0;
	for(int i = 0; i < width; i++){
		for(int j = 0; j < height; j++){
			if(d_res[i*width + j] != h_res[i*width +j]){
				check += 1;
			}
		}
	}
	return check;
}

int main( int argc, char **argv ){
	int width = DEFAULT_WIDTH;
	int height = DEFAULT_HEIGHT;
	if(argc == 3){
		width = atoi(argv[1]);
		height = atoi(argv[2]);
	}
	
	//create vars
	int input1_bytes = height * sizeof(float);
	int num_bytes = width * height * sizeof(float);
	
	//event timers
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	clock_t start_cpu, end_cpu;
	
	//malloc device
	float *d_input1 = (float *) malloc(input1_bytes);
	float *d_input2 = (float *) malloc(num_bytes);
	float *d_result = (float *) malloc(num_bytes);
	
	//malloc host
	float *h_input1 = (float *) malloc(input1_bytes);
	float *h_input2 = (float *) malloc(num_bytes);
	float *h_result = (float *) malloc(num_bytes);
	
	//malloc test
	float *temp = (float *) malloc(width * sizeof(float));
	float *cpu_res = (float *) malloc(num_bytes);
	
	//cuda malloc
	hipMalloc(&d_input1, input1_bytes);
	hipMalloc(&d_input2, num_bytes);
	hipMalloc(&d_result, num_bytes);
	
	//put in data
	for(int o = 0; o < width; o++){
        h_input1[o] = 1;
        for(int p = 0; p < height; p++){
            h_input2[width * o + p] = 1;
        }
    }

	//copy over memory
	hipMemcpy(d_input1, h_input1, input1_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_input2, h_input2, num_bytes, hipMemcpyHostToDevice);
	
	//declare block and grid size for kernel
	int block_size = 32;
	//make grids x y
	int grid_x = (int)ceil((float)width/block_size);
	int grid_y = (int)ceil((float)height/block_size);
	dim3 dim_grid (grid_x, grid_y);
	dim3 dim_block (block_size, block_size);
	
	//start timer 
	hipEventRecord(start);
	
	//run kernel
	unknown_algo<<< dim_grid, dim_block >>> (d_input1, d_input2, d_result, width, height);
	
	//end timer
	hipEventRecord(stop);

	// Copy result back to host
	hipMemcpy(h_result, d_result, num_bytes, hipMemcpyDeviceToHost);
	
	//synchronize https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	//cpu version
	start_cpu = clock();
	cpu_v(h_input1, h_input2, cpu_res, width, height, temp);
	end_cpu = clock();

	int error_count = compare_res(h_result, cpu_res, width, height);
	
	printf("error count: %d\nGPU time for execution: %lf ms\nCPU time for execution: %lf ms\n", error_count, milliseconds, ((float)((end_cpu-start_cpu)*1000))/CLOCKS_PER_SEC);
	printf("gridx: %d, grid y: %d\n", grid_x,grid_y);
	//print error check
	for(int i = 0; i < width; i++){
		printf("%d. ", i);
		for(int j = 0; j < height; j++){
			printf("%d ", (int)h_result[i*width + j]);
		}
		printf("\n");
	}
	for(int i = 0; i < width; i++){
		printf("%d. ", i);
		for(int j = 0; j < height; j++){
			printf("%d ", (int)cpu_res[i*width + j]);
		}
		printf("\n");
	}
	
	//free all vars
	free(h_input1);
	free(h_input2);
	free(h_result);
	free(temp);
	hipFree(d_input1);
	hipFree(d_input2);
	hipFree(d_result);
	
	return 0;
}