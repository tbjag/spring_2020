//optimization homework #4 cs 677 Theodore Jagodits


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include "string.h"
#include <iostream>

#define DEFAULT_SIZE 128
#define TILE_SIZE 16

__global__ void unknown_algo(float *inp1, float *inp2, float *result, int size){
	// make shared
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	float temp = 0.0f;
	for(int j = 0; j < size; j++){
		temp += inp2[id * size + j];
		result[id * size + j] = temp;
		for(int k = 0; k < size; k++){
			//shared input 1 here
			result[id * size + j] += inp1[j] * inp1[k];
		}
	}
}

int main( int argc, char **argv ){
	int size = DEFAULT_SIZE;
	if(argc == 2){
		size = atoi(argv[1]);
	}
	
	//create vars
	int input1_bytes = size * sizeof(float);
	int num_bytes = size * size * sizeof(float);
	
	//event timers
	hipEvent_t start,stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	//malloc device
	float *d_input1 = (float *) malloc(input1_bytes);
	float *d_input2 = (float *) malloc(num_bytes);
	float *d_result = (float *) malloc(num_bytes);
	
	//malloc host
	float *h_input1 = (float *) malloc(input1_bytes);
	float *h_input2 = (float *) malloc(num_bytes);
	float *h_result = (float *) malloc(num_bytes);
	
	//cuda malloc
	hipMalloc(&d_input1, input1_bytes);
	hipMalloc(&d_input2, num_bytes);
	hipMalloc(&d_result, num_bytes);
	
	//put in data
	for(int o = 0; o < size; o++){
        h_input1[o] = 1;
        for(int p = 0; p < size; p++){
            h_input2[size * o + p] = 1;
        }
    }

	//copy over memory
	hipMemcpy(d_input1, h_input1, input1_bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_input2, h_input2, num_bytes, hipMemcpyHostToDevice);
	
	//declare block and grid size for kernel
	int block_size = 128;
	int grid_size = (int)ceil((float)size/block_size);
	
	//start timer 
	hipEventRecord(start);
	
	//run kernel
	unknown_algo<<< grid_size, block_size >>> (d_input1, d_input2, d_result, size);
	
	//end timer
	hipEventRecord(stop);
	
	// Copy result back to host
	hipMemcpy(h_result, d_result, num_bytes, hipMemcpyDeviceToHost);
	
	//synchronize https://devblogs.nvidia.com/how-implement-performance-metrics-cuda-cc/
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	
	//print output
	for(int o = 0; o < size; o++){
        for(int p = 0; p < size; p++){
            printf("%d ", (int)h_result[o*size + p]);
        }
		printf("\n");
    }
	printf("time for execution: %lf ms\n", milliseconds);
	
	//free all vars
	//free(d_input1);
	//free(d_input2);
	//free(d_result);
	free(h_input1);
	free(h_input2);
	free(h_result);
	hipFree(d_input1);
	hipFree(d_input2);
	hipFree(d_result);
	
	return 0;
}